#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>
#include <random>
#include <iomanip>
#include <math.h>
#include <vector>
#include <stdio.h>

__global__ void matrixMultiply(float *A, float *B, float *C, int m, int n, int p) {
    int rowIdx = blockIdx.y * blockDim.y + threadIdx.y;
    int colIdx = blockIdx.x * blockDim.x + threadIdx.x;

    if (rowIdx < m && colIdx < p) {
        // Where we are writing to within C (flattened index)
        int writeMatrixIdx = rowIdx * p + colIdx;

        // Accumulated dot product
        float dotProduct = 0.0;
        
        // Go through n cols in A and n rows in B to compute dot product
        for (int i = 0; i < n; i++) {
            dotProduct += A[rowIdx * n + i] * B[i * p + colIdx];
        }

        C[writeMatrixIdx] = dotProduct;
    }
}

float *compareKernelAndCUBLAS(int m, int n, int p) {
    // std::cout << "Matrix result size: " << m << "x" << p << std::endl;

    float *hA = new float[m * n];
    float *hB = new float[n * p];
    float *hC = new float[m * p];
    float *hCRef = new float[m * p];

    std::random_device rd;
    std::mt19937 gen(rd());

    std::uniform_real_distribution<> dist(1.0, 1000.0);

    for (int i = 0; i < m * n; i++) {
        hA[i] = dist(gen);
    }

    
    for (int i = 0; i < n * p; i++) {
        hB[i] = dist(gen);
    }

    float *dA;
    float *dB;
    float *dC;

    hipMalloc(&dA, m * n * sizeof(float));
    hipMalloc(&dB, n * p * sizeof(float));
    hipMalloc(&dC, m * p * sizeof(float));
    
    hipMemcpy(dA, hA, m * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, n * p * sizeof(float), hipMemcpyHostToDevice);

    int blockSizeRows = 2;
    int blockSizeCols = 2;

    int numBlocksRows = (m + blockSizeRows - 1) / blockSizeRows;
    int numBlocksCols = (p + blockSizeCols - 1) / blockSizeCols;

    dim3 blockSize(blockSizeCols, blockSizeRows);
    dim3 numBlocks(numBlocksCols, numBlocksRows);

    hipEvent_t start;
    hipEvent_t end;

    hipEventCreate(&start);
    hipEventCreate(&end);

    hipEventRecord(start);

    matrixMultiply<<<numBlocks, blockSize>>>(dA, dB, dC, m, n, p);

    hipEventRecord(end);
    hipEventSynchronize(end);

    float kernelMilliseconds = 0;
    hipEventElapsedTime(&kernelMilliseconds, start, end);
    // std::cout << "Kernel time: " << kernelMilliseconds << " ms" << std::endl;  

    hipMemcpy(hC, dC, m * p * sizeof(float), hipMemcpyDeviceToHost);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    const float alpha = 1.0f;
    const float beta =  0.0f;

    hipEventRecord(start);

    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                p, m, n, &alpha,
                dB, p,
                dA, n,
                &beta,
                dC, p);

    hipEventRecord(end);
    hipEventSynchronize(end);

    float cublasMilliseconds;
    hipEventElapsedTime(&cublasMilliseconds, start, end);
    // std::cout << "cuBLAS time: " << cublasMilliseconds << " ms" << std::endl;  

    hipMemcpy(hCRef, dC, m * p * sizeof(float), hipMemcpyDeviceToHost);

    int errors = 0;
    const float relTol = 1e-5f;
    const float absTol = 1e-3f;

    for (int i = 0; i < m * p; ++i) {
        double diff = fabs((double)hCRef[i] - (double)hC[i]);
        double maxVal = fmax(fabs((double)hCRef[i]), fabs((double)hC[i]));

        if (maxVal <= 0) {
            maxVal = 1.0;
        }

        if (diff > absTol && diff / maxVal > relTol) {
            if (++errors <= 10) {
                std::cout << "Mismatch at " << i << ": "
                        << hC[i] << " (yours) vs. " << hCRef[i] << " (cuBLAS)\n";
            }
        }
    }

    if (errors < 10) {
        std::cout << "Results match!" << std::endl;
    }

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    delete[] hA;
    delete[] hB;
    delete[] hC;
    delete[] hCRef;

    return new float[2]{kernelMilliseconds, cublasMilliseconds};
}

int main() {
    /* Run the kernel for each problem size (i.e. square matrix size) and see how much time
       the matrix multiplication takes. Then compute GFLOPS at each problem size.
    */

    FILE *fptr;

    fptr = fopen("data/NaiveMatrixMultVsCuBLAS_GFLOPS.csv", "w");

    fprintf(fptr, "Matrix Size, Kernel GFLOPS, cuBLAS GFLOPS\n");

    for (int problemSize = 32; problemSize <= 2048; problemSize *= 2) {
        float *runtimes = compareKernelAndCUBLAS(problemSize, problemSize, problemSize);

        double naiveMilliseconds = runtimes[0];
        double cublasMilliseconds = runtimes[1];

        double kernelGFLOPS = (2.0 * problemSize * problemSize * problemSize) / (naiveMilliseconds / 1000.0) / 1e9;
        double cublasGFLOPS = (2.0 * problemSize * problemSize * problemSize) / (cublasMilliseconds / 1000.0) / 1e9;

        fprintf(fptr, "%d, %.3f, %.3f\n", problemSize, kernelGFLOPS, cublasGFLOPS);

        delete[] runtimes;
    }

    fclose(fptr);

    system("pause");
    return 0;
}